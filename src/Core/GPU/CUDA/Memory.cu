#include "NNTK/Core/GPU/CUDA/Memory.hpp"

namespace NN
{
namespace internal
{

void
allocate_memory_cuda(void **pointer, size_t bytes)
{ hipMallocManaged(pointer, bytes); }

void
free_memory_cuda(void **pointer)
{ hipFree(pointer); }

void
copy_memory_cuda(void *dst, const void *src, std::size_t count)
{ hipMemcpy(dst, src, count, hipMemcpyDefault); }

} // namespace internal

} // namespace NN
